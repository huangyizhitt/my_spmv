#include "utils.cuh"

__host__ void CudaKernelErrorCheck(const char *prefix, const char *postfix)
{
	if(hipPeekAtLastError() != hipSuccess)
	{
		printf("\n%s%s%s", prefix, hipGetErrorString(hipGetLastError()), postfix);
		hipDeviceReset();
		exit(1);
	}
}
