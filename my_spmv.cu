#include <stdio.h>
#include <hip/hip_runtime.h>


int main(int argc, char **argv)
{
	CmdLine cmd(argc, argv);
	if (args.CheckCmdLineFlag("help"))
	{
		printf(
			"%s "
            "[--csrmv | --hybmv | --bsrmv ] "
            "[--device=<device-id>] "
            "[--quiet] "
            "[--v] "
            "[--i=<timing iterations>] "
            "[--fp32] "
            "[--alpha=<alpha scalar (default: 1.0)>] "
            "[--beta=<beta scalar (default: 0.0)>] "
            "\n\t"
                "--mtx=<matrix market file> "
            "\n\t"
                "--dense=<cols>"
            "\n\t"
                "--grid2d=<width>"
            "\n\t"
                "--grid3d=<width>"
            "\n\t"
                "--wheel=<spokes>"
            "\n", argv[0]);
		exit(0);
	}
}
